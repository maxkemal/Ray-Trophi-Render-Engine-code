#include "hip/hip_runtime.h"
extern "C" __global__ void __raygen__rg() {
    // Example ray origin and direction
    float3 ray_origin = make_float3(0.0f, 0.0f, 0.0f);
    float3 ray_direction = make_float3(0.0f, 0.0f, -1.0f);

    // Define tmin and tmax
    float tmin = 0.0f;
    float tmax = 1.0e16f;

    // Trace the ray
    optixTrace(
        pipeline,
        ray_origin,
        ray_direction,
        tmin,
        tmax,
        0, // Ray type (customize as needed)
        OptixVisibilityMask(1),
        OPTIX_RAY_FLAG_NONE,
        0, // SBT offset
        1, // SBT stride
        0, // Miss program index
        0, // Hit program index
        0  // Any hit program index
    );
}
