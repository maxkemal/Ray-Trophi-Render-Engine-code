#include "hip/hip_runtime.h"
// ... SBT olu�turma k�sm� ...
OptixShaderBindingTable sbt;
// ... SBT'yi konfig�re etme

// Hitgroup'ta sonu�lar� depola
extern "C" __global__ void __closesthit__radiance() {
    const HitGroupData* hitData = reinterpret_cast<HitGroupData*>(optixGetSbtData());
    // ... Di�er hesaplamalar

    // Sonucu SBT'nin payload b�l�m�ne yaz
    OptixHitResult* hitResult = reinterpret_cast<OptixHitResult*>(optixGetSbtDataPayload());
    hitResult->hasHit = true;
    // ... Di�er sonu�lar� ayarla
}

// Host'ta sonu�lar� oku
OptixHitResult* hostResult = new OptixHitResult[numRays];
hipMemcpy(hostResult, sbt.payloadBase, numRays * sizeof(OptixHitResult), hipMemcpyDeviceToHost);